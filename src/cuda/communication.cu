#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/communication.hpp"

namespace duckdb {

// Define the host function that launches the CUDA kernel
int* sendDataToGPU(int* data, int size) {
    printf("Send data to GPU\n");
    // use cudamemcpy
    int** target = new int*[1];
    hipMalloc((void**) &target[0], size * sizeof(int));
    hipMemcpy(target, data, size * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("Done sending data to GPU\n");
    return target[0];
}

} // namespace duckdb