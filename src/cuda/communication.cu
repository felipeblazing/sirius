#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "communication.hpp"
#include "gpu_columns.hpp"
#include "operator/cuda_helper.cuh"

namespace duckdb {

template void
callCudaMemcpyHostToDevice<int>(int* dest, int* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<uint64_t>(uint64_t* dest, uint64_t* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<float>(float* dest, float* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<double>(double* dest, double* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<uint8_t>(uint8_t* dest, uint8_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<int>(int* dest, int* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<uint64_t>(uint64_t* dest, uint64_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<float>(float* dest, float* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<double>(double* dest, double* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<uint8_t>(uint8_t* dest, uint8_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<char>(char* dest, char* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<string_t>(string_t* dest, string_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToDevice<uint8_t>(uint8_t* dest, uint8_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToDevice<int>(int* dest, int* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToDevice<uint64_t>(uint64_t* dest, uint64_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToDevice<float>(float* dest, float* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToDevice<double>(double* dest, double* src, size_t size, int gpu);

template <typename T> 
void callCudaMemcpyHostToDevice(T* dest, T* src, size_t size, int gpu) {
    CHECK_ERROR();
    if (size == 0) {
        printf("N is 0\n");
        return;
    }
    printf("Send data to GPU\n");
    hipSetDevice(gpu);
    gpuErrchk(hipMemcpy(dest, src, size * sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());
    hipSetDevice(0);
    printf("Done sending data to GPU\n");
}

template <typename T> 
void callCudaMemcpyDeviceToHost(T* dest, T* src, size_t size, int gpu) {
    CHECK_ERROR();
    if (size == 0) {
        printf("N is 0\n");
        return;
    }
    SETUP_TIMING();
    START_TIMER();
    printf("Send data to CPU\n");
    hipSetDevice(gpu);
    printf("Transferred bytes: %ld\n", size * sizeof(T));
    if (src == nullptr) {
        printf("src is null\n");
    }
    if (dest == nullptr) {
        printf("dest is null\n");
    }
    gpuErrchk(hipMemcpy(dest, src, size * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_ERROR();
    gpuErrchk(hipDeviceSynchronize());
    hipSetDevice(0);
    printf("Done sending data to CPU\n");
    STOP_TIMER();
}

template <typename T> 
void callCudaMemcpyDeviceToDevice(T* dest, T* src, size_t size, int gpu) {
    CHECK_ERROR();
    if (size == 0) {
        printf("N is 0\n");
        return;
    }
    SETUP_TIMING();
    START_TIMER();
    printf("Send data within GPU\n");
    hipSetDevice(gpu);
    printf("Transferred bytes: %ld\n", size * sizeof(T));
    if (src == nullptr) {
        printf("src is null\n");
    }
    if (dest == nullptr) {
        printf("dest is null\n");
    }
    gpuErrchk(hipMemcpy(dest, src, size * sizeof(T), hipMemcpyDeviceToDevice));
    CHECK_ERROR();
    gpuErrchk(hipDeviceSynchronize());
    hipSetDevice(0);
    printf("Done sending data to GPU\n");
    STOP_TIMER();
}

// Define the host function that launches the CUDA kernel
int* sendDataToGPU(int* data, int size) {
    printf("Send data to GPU\n");
    // use cudamemcpy
    int* target;
    hipMalloc((void**) &target, size * sizeof(int));
    hipMemcpy(target, data, size * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("Done sending data to GPU\n");
    return target;
}

} // namespace duckdb