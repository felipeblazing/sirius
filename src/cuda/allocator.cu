#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "gpu_buffer_manager.hpp"

namespace duckdb {

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template int*
callCudaMalloc<int>(size_t size, int gpu);

template uint64_t*
callCudaMalloc<uint64_t>(size_t size, int gpu);

template uint8_t*
callCudaMalloc<uint8_t>(size_t size, int gpu);

template float*
callCudaMalloc<float>(size_t size, int gpu);

template double*
callCudaMalloc<double>(size_t size, int gpu);

template void
callCudaFree<int>(int* ptr, int gpu);

template void
callCudaFree<uint64_t>(uint64_t* ptr, int gpu);

template void
callCudaFree<uint8_t>(uint8_t* ptr, int gpu);

template void
callCudaFree<float>(float* ptr, int gpu);

template void
callCudaFree<double>(double* ptr, int gpu);

template <typename T>
T* callCudaMalloc(size_t size, int gpu) {
    T* ptr;
    hipSetDevice(gpu);
    gpuErrchk(hipMalloc((void**) &ptr, size * sizeof(T)));
    hipDeviceSynchronize();
    hipSetDevice(0);
    return ptr;
}

template <typename T>
void callCudaFree(T* ptr, int gpu) {
    hipSetDevice(gpu);
    gpuErrchk(hipFree(ptr));
    hipDeviceSynchronize();
    hipSetDevice(gpu);
}

} // namespace duckdb