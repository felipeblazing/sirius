#include "cudf/cudf_utils.hpp"
#include "gpu_physical_ungrouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"
namespace duckdb {

template <cudf::reduce_aggregation::Kind kind>
static std::unique_ptr<cudf::reduce_aggregation> make_reduce_aggregation()
{
  switch (kind) {
    case cudf::reduce_aggregation::MIN:
      return cudf::make_min_aggregation<cudf::reduce_aggregation>();
    case cudf::reduce_aggregation::MAX:
      return cudf::make_max_aggregation<cudf::reduce_aggregation>();
    case cudf::reduce_aggregation::MEAN:
      return cudf::make_mean_aggregation<cudf::reduce_aggregation>();
    case cudf::reduce_aggregation::SUM:
      return cudf::make_sum_aggregation<cudf::reduce_aggregation>();
    default:
      throw NotImplementedException("Unsupported reduce aggregation");
  }
}

void cudf_aggregate(vector<shared_ptr<GPUColumn>>& column, uint64_t num_aggregates, AggregationType* agg_mode) 
{
    if (column[0]->column_length == 0) {
        printf("N is 0\n");
        for (int agg_idx = 0; agg_idx < num_aggregates; agg_idx++) {
            if (agg_mode[agg_idx] == AggregationType::COUNT_STAR || agg_mode[agg_idx] == AggregationType::COUNT) {
                column[agg_idx] = make_shared_ptr<GPUColumn>(0, ColumnType::INT64, column[agg_idx]->data_wrapper.data);
            } else {
                column[agg_idx] = make_shared_ptr<GPUColumn>(0, column[agg_idx]->data_wrapper.type, column[agg_idx]->data_wrapper.data);
            }
        }
        return;
    }

    printf("CUDF Aggregate\n");

    GPUBufferManager *gpuBufferManager = &(GPUBufferManager::GetInstance());
    cudf::set_current_device_resource(gpuBufferManager->mr);

    uint64_t size = 0;
    for (int agg = 0; agg < num_aggregates; agg++) {
        if (column[agg]->data_wrapper.data != nullptr) {
            size = column[agg]->column_length;
            break;
        }
    }

    for (int agg = 0; agg < num_aggregates; agg++) {
        if (column[agg]->data_wrapper.data == nullptr && agg_mode[agg] == AggregationType::COUNT && column[agg]->column_length == 0) {
            uint64_t* temp = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
            hipMemset(temp, 0, sizeof(uint64_t));
            column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT64, reinterpret_cast<uint8_t*>(temp));
        } else if (column[agg]->data_wrapper.data == nullptr && agg_mode[agg] == AggregationType::SUM && column[agg]->column_length == 0) {
            uint64_t* temp = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
            hipMemset(temp, 0, sizeof(uint64_t));
            column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT64, reinterpret_cast<uint8_t*>(temp));
        } else if (column[agg]->data_wrapper.data == nullptr && agg_mode[agg] == AggregationType::COUNT_STAR && column[agg]->column_length != 0) {
            uint64_t* res = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
            res[0] = size;
            uint64_t* result_temp = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
            hipMemcpy(result_temp, res, sizeof(uint64_t), hipMemcpyHostToDevice);
            column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT64, reinterpret_cast<uint8_t*>(result_temp));
        } else if (agg_mode[agg] == AggregationType::SUM) {
            auto aggregate = make_reduce_aggregation<cudf::reduce_aggregation::SUM>();
            auto cudf_column = column[agg]->convertToCudfColumn();
            auto result = cudf::reduce(cudf_column, *aggregate, cudf_column.type());
            column[agg]->setFromCudfScalar(*result, gpuBufferManager);
        } else if (agg_mode[agg] == AggregationType::AVERAGE) {
            auto aggregate = make_reduce_aggregation<cudf::reduce_aggregation::MEAN>();
            auto cudf_column = column[agg]->convertToCudfColumn();
            auto result = cudf::reduce(cudf_column, *aggregate, cudf_column.type());
            column[agg]->setFromCudfScalar(*result, gpuBufferManager);
        } else if (agg_mode[agg] == AggregationType::MIN) {
            auto aggregate = make_reduce_aggregation<cudf::reduce_aggregation::MIN>();
            auto cudf_column = column[agg]->convertToCudfColumn();
            auto result = cudf::reduce(cudf_column, *aggregate, cudf_column.type());
            column[agg]->setFromCudfScalar(*result, gpuBufferManager);
        } else if (agg_mode[agg] == AggregationType::MAX) {
            auto aggregate = make_reduce_aggregation<cudf::reduce_aggregation::MAX>();
            auto cudf_column = column[agg]->convertToCudfColumn();
            auto result = cudf::reduce(cudf_column, *aggregate, cudf_column.type());
            column[agg]->setFromCudfScalar(*result, gpuBufferManager);
        } else if (agg_mode[agg] == AggregationType::COUNT) {
            uint64_t* res = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
            res[0] = size;
            uint64_t* result_temp = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
            hipMemcpy(result_temp, res, sizeof(uint64_t), hipMemcpyHostToDevice);
            column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT64, reinterpret_cast<uint8_t*>(result_temp));
        } else if (agg_mode[agg] == AggregationType::FIRST) {
            if (column[agg]->data_wrapper.type == ColumnType::INT64) {
                uint64_t* result_temp = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<uint64_t*>(column[agg]->data_wrapper.data), sizeof(uint64_t), hipMemcpyDeviceToDevice);
                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT64, reinterpret_cast<uint8_t*>(result_temp));
            } else if (column[agg]->data_wrapper.type == ColumnType::INT32) {
                int32_t* result_temp = gpuBufferManager->customCudaMalloc<int32_t>(1, 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<int32_t*>(column[agg]->data_wrapper.data), sizeof(int32_t), hipMemcpyDeviceToDevice);
                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::INT32, reinterpret_cast<uint8_t*>(result_temp));
            } else if (column[agg]->data_wrapper.type == ColumnType::FLOAT32) {
                float* result_temp = gpuBufferManager->customCudaMalloc<float>(1, 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<float*>(column[agg]->data_wrapper.data), sizeof(float), hipMemcpyDeviceToDevice);
                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::FLOAT32, reinterpret_cast<uint8_t*>(result_temp));
            } else if (column[agg]->data_wrapper.type == ColumnType::FLOAT64) {
                double* result_temp = gpuBufferManager->customCudaMalloc<double>(1, 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<double*>(column[agg]->data_wrapper.data), sizeof(double), hipMemcpyDeviceToDevice);
                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::FLOAT64, reinterpret_cast<uint8_t*>(result_temp));
            } else if (column[agg]->data_wrapper.type == ColumnType::BOOLEAN) {
                uint8_t* result_temp = gpuBufferManager->customCudaMalloc<uint8_t>(1, 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<uint8_t*>(column[agg]->data_wrapper.data), sizeof(uint8_t), hipMemcpyDeviceToDevice);
                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::BOOLEAN, reinterpret_cast<uint8_t*>(result_temp));
            } else if (column[agg]->data_wrapper.type == ColumnType::VARCHAR) {
                uint64_t* length = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
                hipMemcpy(length, column[agg]->data_wrapper.offset + 1, sizeof(uint64_t), hipMemcpyDeviceToHost);

                char* result_temp = gpuBufferManager->customCudaMalloc<char>(length[0], 0, 0);
                hipMemcpy(result_temp, reinterpret_cast<char*>(column[agg]->data_wrapper.data), length[0], hipMemcpyDeviceToDevice);

                uint64_t* new_offset = gpuBufferManager->customCudaMalloc<uint64_t>(2, 0, 0);
                hipMemcpy(new_offset, column[agg]->data_wrapper.offset, 2 * sizeof(uint64_t), hipMemcpyDeviceToDevice);

                column[agg] = make_shared_ptr<GPUColumn>(1, ColumnType::VARCHAR, reinterpret_cast<uint8_t*>(result_temp), new_offset, length[0], true);
            }
        } 
        else {
            throw NotImplementedException("Aggregate function not supported");
        }
    }

}

} //namespace duckdb