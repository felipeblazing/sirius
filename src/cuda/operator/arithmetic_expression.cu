#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_expression_executor.hpp"

namespace duckdb {

template <typename T, int B, int I>
__global__ void binary_expression(T *a, T *b, T *result, uint64_t N, int op_mode) {
    
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            // printf("Thread %d: Processing element %ld\n", threadIdx.x, offset);
            // printf("Addition: %.2f + %.2f\n", a[offset], b[offset]);
            if (op_mode == 0) {
                // printf("I'm here\n");
                // printf("Addition: %.2f + %.2f\n", a[offset], b[offset]);
                result[offset] = a[offset] + b[offset];
                // printf("Addition: %.2f + %.2f = %.2f\n", a[offset], b[offset], result[offset]);
            } else {
                cudaAssert(0);
            }
        }
    }
}

__global__ void test(double *b, double *result, uint64_t N, int op_mode) {
    
    uint64_t tile_size = 128 * 4;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    // // Ensure that the memory access is aligned
    // if (reinterpret_cast<uintptr_t>(b) % alignof(double) == 0) {
    //     printf("Memory is not properly aligned 1\n");
    //     // return;
    // } else if (reinterpret_cast<uintptr_t>(b) % alignof(int) == 0) {
    //     printf("Memory is not properly aligned 2\n");
    //     // return;
    // } else if (reinterpret_cast<uintptr_t>(b) % alignof(char) == 0) {
    //     printf("Memory is not properly aligned 3\n");
    //     // return;
    // }

    printf("%.2f\n", (reinterpret_cast<double*>(b))[0]);

    // #pragma unroll
    // for (int ITEM = 0; ITEM < 4; ++ITEM) {
    //     if (threadIdx.x + ITEM * 128 < num_tile_items) {
    //         int offset = tile_offset + threadIdx.x + ITEM * 128;
    //         printf("Thread %d: Processing element %ld\n", threadIdx.x, offset);
    //         printf("%.2f %.2f\n", a[0], b[0]);
    //     }
    // }
}

// template
// __global__ void binary_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(int *a, int *b, int *result, uint64_t N, int op_mode);
// template
// __global__ void binary_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t *a, uint64_t *b, uint64_t *result, uint64_t N, int op_mode);
// template
// __global__ void binary_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(float *a, float *b, float *result, uint64_t N, int op_mode);
template
__global__ void binary_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(double *a, double *b, double *result, uint64_t N, int op_mode);
// template
// __global__ void binary_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(uint8_t *a, uint8_t *b, uint8_t *result, uint64_t N, int op_mode);

// Define the host function that launches the CUDA kernel
template <typename T>
void binaryExpression(T *a, T *b, T *result, uint64_t N, int op_mode) {
    printf("Launching Binary Expression Kernel\n");
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    // CHECK_ERROR();
    // // printf("%d\n", (N  tile_items - 1)/tile_items);
    // double* host_data_b = new double[N];
    // hipMemcpy(host_data_b, b, N * sizeof(double), hipMemcpyDeviceToHost);
    // // for (int i = 0; i < 10; i++) {
    // //     printf("%f ", reinterpret_cast<double*>(host_data_b)[i]);
    // // }
    // // printf("\n");
    // CHECK_ERROR();

    // double* new_b;
    // hipMalloc((void**) &new_b, sizeof(double) * N);
    // CHECK_ERROR();
    // hipMemcpy(new_b, host_data_b, N * sizeof(double), hipMemcpyHostToDevice);
    // printf("\n");
    // CHECK_ERROR();
    if (reinterpret_cast<uintptr_t>(a) % alignof(double) != 0) {
        printf("Memory is not properly aligned\n");
    }
    if (reinterpret_cast<uintptr_t>(b) % alignof(double) != 0) {
        printf("Memory is not properly aligned\n");
    }
    if (reinterpret_cast<uintptr_t>(result) % alignof(double) != 0) {
        printf("Memory is not properly aligned\n");
    }
    binary_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, result, N, op_mode);
    // test<<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(b, result, N, op_mode);
    CHECK_ERROR();
    hipDeviceSynchronize();
}

// template
// void binaryExpression<int>(int *a, int *b, int *result, uint64_t N, int op_mode);
// template
// void binaryExpression<uint64_t>(uint64_t *a, uint64_t *b, uint64_t *result, uint64_t N, int op_mode);
// template
// void binaryExpression<float>(float *a, float *b, float *result, uint64_t N, int op_mode);
template
void binaryExpression<double>(double *a, double *b, double *result, uint64_t N, int op_mode);
// template
// void binaryExpression<uint8_t>(uint8_t *a, uint8_t *b, uint8_t *result, uint64_t N, int op_mode);

} // namespace duckdb