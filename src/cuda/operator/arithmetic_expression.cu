#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_expression_executor.hpp"

namespace duckdb {

template <typename T, int B, int I>
__global__ void binary_expression(T *a, T *b, T *result, uint64_t N, int op_mode) {
    
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            if (op_mode == 0) {
                result[offset] = a[offset] * (1 - b[offset]);
            } else {
                cudaAssert(0);
            }
        }
    }
}

template
__global__ void binary_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(int *a, int *b, int *result, uint64_t N, int op_mode);
template
__global__ void binary_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t *a, uint64_t *b, uint64_t *result, uint64_t N, int op_mode);
template
__global__ void binary_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(float *a, float *b, float *result, uint64_t N, int op_mode);
template
__global__ void binary_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(double *a, double *b, double *result, uint64_t N, int op_mode);
template
__global__ void binary_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(uint8_t *a, uint8_t *b, uint8_t *result, uint64_t N, int op_mode);

// Define the host function that launches the CUDA kernel
template <typename T>
void binaryExpression(T *a, T *b, T *result, uint64_t N, int op_mode) {
    printf("Launching Binary Expression Kernel\n");
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    binary_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, result, N, op_mode);
    CHECK_ERROR();
    hipDeviceSynchronize();
}

template
void binaryExpression<int>(int *a, int *b, int *result, uint64_t N, int op_mode);
template
void binaryExpression<uint64_t>(uint64_t *a, uint64_t *b, uint64_t *result, uint64_t N, int op_mode);
template
void binaryExpression<float>(float *a, float *b, float *result, uint64_t N, int op_mode);
template
void binaryExpression<double>(double *a, double *b, double *result, uint64_t N, int op_mode);
template
void binaryExpression<uint8_t>(uint8_t *a, uint8_t *b, uint8_t *result, uint64_t N, int op_mode);

} // namespace duckdb