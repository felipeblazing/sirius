#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_expression_executor.hpp"

namespace duckdb {

// string t = "((P_BRAND != 45) AND ((P_TYPE < 65) OR (P_TYPE >= 70)) AND (P_SIZE IN (49, 14, 23, 45, 19, 3, 36, 9)))";
// string t = "((L_COMMITDATE < L_RECEIPTDATE) AND (L_SHIPDATE < L_COMMITDATE) AND (L_SHIPMODE IN (4, 6)))";
// string t = "(((P_BRAND = 12) AND (L_QUANTITY <= 11) AND (P_SIZE <= 5) AND (P_CONTAINER IN (0, 1, 4, 5))) OR ((P_BRAND = 23) AND (L_QUANTITY >= 10) AND (L_QUANTITY <= 20) AND (P_SIZE <= 10) AND (P_CONTAINER IN (17, 18, 20, 21))) OR ((P_BRAND = 34) AND (L_QUANTITY >= 20) AND (L_QUANTITY <= 30) AND (P_SIZE <= 15) AND (P_CONTAINER IN (8, 9, 12, 13))))";
// string t = "(((N_NATIONKEY = 6) AND (N_NATIONKEY = 7)) OR ((N_NATIONKEY = 7) AND (N_NATIONKEY = 6)))";
// string t = "(((P_TYPE + 3) % 5) = 0)";

template <typename T, int B, int I>
__global__ void comparison_expression(const T *a, const T *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count) {

    typedef hipcub::BlockScan<int, B> BlockScanInt;

    __shared__ union TempStorage
    {
        typename BlockScanInt::TempStorage scan;
    } temp_storage;

    T items_a[I];
    T items_b[I];
    int selection_flags[I];

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    int t_count = 0; // Number of items selected per thread
    int c_t_count = 0; //Prefix sum of t_count
    __shared__ uint64_t block_off;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        selection_flags[ITEM] = 0;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            items_a[ITEM] = a[tile_offset + threadIdx.x + ITEM * B];
            items_b[ITEM] = b[tile_offset + threadIdx.x + ITEM * B];
            if (compare_mode == 0) {
                selection_flags[ITEM] = (items_a[ITEM] == items_b[ITEM]);
            } else if (compare_mode == 1) {
                selection_flags[ITEM] = (items_a[ITEM] != items_b[ITEM]);
            } else if (compare_mode == 2) {
                selection_flags[ITEM] = (items_a[ITEM] > items_b[ITEM]);
            } else if (compare_mode == 3) {
                selection_flags[ITEM] = (items_a[ITEM] >= items_b[ITEM]);
            } else if (compare_mode == 4) {
                selection_flags[ITEM] = (items_a[ITEM] < items_b[ITEM]);
            } else if (compare_mode == 5) {
                selection_flags[ITEM] = (items_a[ITEM] <= items_b[ITEM]);
            } else {
                cudaAssert(0);
            }
            if(selection_flags[ITEM]) t_count++;
        }
    }

    //Barrier
    __syncthreads();

    BlockScanInt(temp_storage.scan).ExclusiveSum(t_count, c_t_count); //doing a prefix sum of all the previous threads in the block and store it to c_t_count
    if(threadIdx.x == blockDim.x - 1) { //if the last thread in the block, add the prefix sum of all the prev threads + sum of my threads to global variable total
        block_off = atomicAdd(count, (unsigned long long) t_count+c_t_count); //the previous value of total is gonna be assigned to block_off
    } //block_off does not need to be global (it's just need to be shared), because it will get the previous value from total which is global

    __syncthreads();

    if (is_count) return;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            if(selection_flags[ITEM]) {
                uint64_t offset = block_off + c_t_count++;
                row_ids[offset] = tile_offset + threadIdx.x + ITEM * B;
            }
        }
    }
}

template <typename T, int B, int I>
__global__ void comparison_constant_expression(const T *a, const T b, const T c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count) {

    typedef hipcub::BlockScan<int, B> BlockScanInt;

    __shared__ union TempStorage
    {
        typename BlockScanInt::TempStorage scan;
    } temp_storage;

    T items_a[I];
    int selection_flags[I];

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    int t_count = 0; // Number of items selected per thread
    int c_t_count = 0; //Prefix sum of t_count
    __shared__ uint64_t block_off;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        selection_flags[ITEM] = 0;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            items_a[ITEM] = a[tile_offset + threadIdx.x + ITEM * B];
            if (compare_mode == 0) {
                selection_flags[ITEM] = (items_a[ITEM] == b);
            } else if (compare_mode == 1) {
                selection_flags[ITEM] = (items_a[ITEM] != b);
            } else if (compare_mode == 2) {
                selection_flags[ITEM] = (items_a[ITEM] > b);
            } else if (compare_mode == 3) {
                selection_flags[ITEM] = (items_a[ITEM] >= b);
            } else if (compare_mode == 4) {
                selection_flags[ITEM] = (items_a[ITEM] < b);
            } else if (compare_mode == 5) {
                selection_flags[ITEM] = (items_a[ITEM] <= b);
            } else if (compare_mode == 6) {
                selection_flags[ITEM] = ((items_a[ITEM] >= b) && (items_a[ITEM] <= c));
            } else if (compare_mode == 7) {
                selection_flags[ITEM] = ((items_a[ITEM] < b) || (items_a[ITEM] > c));
            } else {
                cudaAssert(0);
            }
            if(selection_flags[ITEM]) {
                t_count++;
            }
        }
    }

    //Barrier
    __syncthreads();

    BlockScanInt(temp_storage.scan).ExclusiveSum(t_count, c_t_count); //doing a prefix sum of all the previous threads in the block and store it to c_t_count
    if(threadIdx.x == blockDim.x - 1) { //if the last thread in the block, add the prefix sum of all the prev threads + sum of my threads to global variable total
        block_off = atomicAdd(count, (unsigned long long) t_count+c_t_count); //the previous value of total is gonna be assigned to block_off
    } //block_off does not need to be global (it's just need to be shared), because it will get the previous value from total which is global

    __syncthreads();

    if (is_count) return;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            if(selection_flags[ITEM]) {
                uint64_t offset = block_off + c_t_count++;
                row_ids[offset] = tile_offset + threadIdx.x + ITEM * B;
            }
        }
    }
}

template<typename T>
__global__ void test(T* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < 100; i++) {
            printf("%.2f ", a[i]);
        }
        printf("\n");
    }
}

template
__global__ void comparison_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(const int *a, const int *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, const uint64_t *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(const float *a, const float *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, const double *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint8_t *a, const uint8_t *b, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);

template
__global__ void comparison_constant_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(const int *a, const int b, const int c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_constant_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, const uint64_t b, const uint64_t c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_constant_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(const float *a, const float b, const float c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_constant_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, const double b, const double c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);
template
__global__ void comparison_constant_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint8_t *a, const uint8_t b, const uint8_t c, uint64_t *row_ids, unsigned long long* count, uint64_t N, int compare_mode, int is_count);

template <typename T>
void comparisonConstantExpression(T *a, T b, T c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode) {
    CHECK_ERROR();
    if (N == 0) {
        uint64_t* h_count = new uint64_t[1];
        h_count[0] = 0;
        count = h_count;
        printf("N is 0\n");
        return;
    }
    printf("Launching Comparison Expression Kernel\n");
    hipMemset(count, 0, sizeof(uint64_t));
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    comparison_constant_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, c, row_ids, (unsigned long long*) count, N, op_mode, 1);
    CHECK_ERROR();
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    uint64_t* h_count = new uint64_t[1];
    hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
    row_ids = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
    hipMemset(count, 0, sizeof(uint64_t));
    comparison_constant_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, c, row_ids, (unsigned long long*) count, N, op_mode, 0);
    CHECK_ERROR();
    hipDeviceSynchronize();
    count = h_count;
    printf("Count: %lu\n", h_count[0]);
}

template <typename T>
void comparisonExpression(T *a, T *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode) {
    CHECK_ERROR();
    if (N == 0) {
        uint64_t* h_count = new uint64_t[1];
        h_count[0] = 0;
        count = h_count;
        printf("N is 0\n");
        return;
    }
    printf("Launching Comparison Expression Kernel\n");
    hipMemset(count, 0, sizeof(uint64_t));
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    comparison_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, row_ids, (unsigned long long*) count, N, op_mode, 1);
    CHECK_ERROR();
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    uint64_t* h_count = new uint64_t[1];
    hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
    row_ids = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
    hipMemset(count, 0, sizeof(uint64_t));
    comparison_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, b, row_ids, (unsigned long long*) count, N, op_mode, 0);
    CHECK_ERROR();
    hipDeviceSynchronize();
    count = h_count;
    printf("Count: %lu\n", h_count[0]);
}

template
void comparisonConstantExpression<int>(int *a, int b, int c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonConstantExpression<uint64_t>(uint64_t *a, uint64_t b, uint64_t c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonConstantExpression<float>(float *a, float b, float c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonConstantExpression<double>(double *a, double b, double c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonConstantExpression<uint8_t>(uint8_t *a, uint8_t b, uint8_t c, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);


template
void comparisonExpression<int>(int *a, int *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonExpression<uint64_t>(uint64_t *a, uint64_t *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonExpression<float>(float *a, float *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonExpression<double>(double *a, double *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);
template
void comparisonExpression<uint8_t>(uint8_t *a, uint8_t *b, uint64_t* &row_ids, uint64_t* &count, uint64_t N, int op_mode);

} // namespace duckdb