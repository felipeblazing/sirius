#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_columns.hpp"

namespace duckdb {

template <typename T, int B, int I>
__global__ void materialize_expression(const T *a, T* result, uint64_t *row_ids, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            int items_ids = row_ids[tile_offset + threadIdx.x + ITEM * B];
            result[tile_offset + threadIdx.x + ITEM * B] = a[items_ids];
            // cudaAssert(a[items_ids] == 19940101);
            // printf("Result: %ld\n", result[tile_offset + threadIdx.x + ITEM * B]);
        }
    }

}

template <typename T>
__global__ void test(T* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < 100; i++) {
            printf("%ld ", a[i]);
        }
        printf("\n");
    }
}

template
__global__ void materialize_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(const int *a, int* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(const float *a, float* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, double* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint8_t *a, uint8_t* result, uint64_t *row_ids, uint64_t N);

template
__global__ void test<uint64_t>(uint64_t* a, uint64_t N);
template
__global__ void test<double>(double* a, uint64_t N);
template
__global__ void test<int>(int* a, uint64_t N);
template
__global__ void test<float>(float* a, uint64_t N);
template
__global__ void test<uint8_t>(uint8_t* a, uint64_t N);

template <typename T>
void materializeExpression(T *a, T* result, uint64_t *row_ids, uint64_t N) {
    printf("Launching Materialize Kernel\n");
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    // printf("N: %ld\n", N);
    // test<T><<<1, 1>>>(a, N);
    // test<uint64_t><<<1, 1>>>(row_ids, N);
    materialize_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, result, row_ids, N);
    CHECK_ERROR();
    // test<T><<<1, 1>>>(result, N);
    hipDeviceSynchronize();
}

template
void materializeExpression<int>(int *a, int* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<uint64_t>(uint64_t *a, uint64_t* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<float>(float *a, float* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<double>(double *a, double* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<uint8_t>(uint8_t *a, uint8_t* result, uint64_t *row_ids, uint64_t N);

} // namespace duckdb